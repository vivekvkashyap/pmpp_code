#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define CHANNELS 3
#define BLUR_SIZE 4

__global__ void blur_gpu(unsigned char *d_blur, unsigned char *d_color, int width, int height, int channels) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        for (int c = 0; c < channels; c++) {
            int pixval = 0;
            int pixels = 0;

            for (int i = -BLUR_SIZE; i <= BLUR_SIZE; i++) {
                for (int j = -BLUR_SIZE; j <= BLUR_SIZE; j++) {
                    int curRow = row + i;
                    int curCol = col + j;

                    if (curRow >= 0 && curRow < height && curCol >= 0 && curCol < width) {
                        int pixel_index = (curRow * width + curCol) * channels + c;
                        pixval += d_color[pixel_index];
                        ++pixels;
                    }
                }
            }

            int out_index = (row * width + col) * channels + c;
            d_blur[out_index] = (unsigned char)(pixval / pixels);
        }
    }
}

void save_image_jpg(unsigned char* image_data, int width, int height, int channels, const char* filename) {
    int result = stbi_write_jpg(filename, width, height, channels, image_data, 90); 
    if (result) {
        printf("Image saved successfully: %s\n", filename);
    } else {
        printf("Failed to save image\n");
    }
}

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main() {
    unsigned char *h_blur_gpu;
    unsigned char *d_color, *d_blur;

    int width, height, channels;
    unsigned char *h_color = stbi_load("f1_ferrari.jpg", &width, &height, &channels, 0);
    if (!h_color) {
        printf("Failed to load image\n");
        return -1;
    }

    printf("Image loaded: %d x %d pixels, %d channels\n", width, height, channels);

    size_t color_size = width * height * channels * sizeof(unsigned char);

    h_blur_gpu = (unsigned char*)malloc(color_size);

    hipMalloc(&d_color, color_size);
    hipMalloc(&d_blur, color_size);

    hipMemcpy(d_color, h_color, color_size, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16, 1);
    dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x,
                 (height + dimBlock.y - 1) / dimBlock.y,
                 1);

    printf("Performing warmup runs...\n");
    for (int i = 0; i < 3; i++) {
        blur_gpu<<<dimGrid, dimBlock>>>(d_blur, d_color, width, height, channels);
        hipDeviceSynchronize();
    }

    printf("Benchmarking GPU implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        blur_gpu<<<dimGrid, dimBlock>>>(d_blur, d_color, width, height, channels);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_total_time += (end_time - start_time);
    }
    double gpu_avg_time = gpu_total_time / 20.0;
    printf("GPU avg time: %.3f milliseconds\n", gpu_avg_time * 1000);

    hipMemcpy(h_blur_gpu, d_blur, color_size, hipMemcpyDeviceToHost);

    save_image_jpg(h_blur_gpu, width, height, 3, "output_gray.jpg");

    stbi_image_free(h_color);
    free(h_blur_gpu);
    hipFree(d_color);
    hipFree(d_blur);

    return 0;
}
