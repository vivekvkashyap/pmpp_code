#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define N 256
#define TILE_WIDTH 16

void matrix_mul_cpu(float *A, float *B, float *C, int n){
    for (int i=0; i<n; i++){
        for (int j=0; j<n; j++){
            float sum = 0.0f;
            for (int k=0; k<n; k++){
                sum += A[i * n + k] * B[k * n + j];
            }
            C[i * n + j] = sum;
        }
    }
}

__global__ void matrix_mul_shared_mem(float *A, float *B, float *C, int n){
    __shared__ float A_d[TILE_WIDTH][TILE_WIDTH];
    __shared__ float B_d[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float Pval = 0.0;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    for (int ph=0; ph<n/TILE_WIDTH; ph++){
        A_d[ty][tx] = A[Row*n + ph*TILE_WIDTH + tx];
        B_d[ty][tx] = B[(ph*TILE_WIDTH + ty)*n + Col];
        __syncthreads();

        for (int k=0; k<TILE_WIDTH; k++){
            Pval += A_d[ty][k] * B_d[k][tx];
        }
        __syncthreads();
    }
    C[Row * n + Col] = Pval;
}


void init_matrix(float *mat, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            mat[i * cols + j] = (float)rand() / RAND_MAX;
        }
    }
}

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main(){
    float *h_A, *h_B, *h_C_cpu, *h_C_gpu;
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C_cpu = (float*)malloc(size);
    h_C_gpu = (float*)malloc(size);

    srand(time(NULL));
    init_matrix(h_A, N, N);
    init_matrix(h_B, N, N);

    memset(h_C_cpu, 0, size);
    memset(h_C_gpu, 0, size);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C_gpu, size, hipMemcpyHostToDevice);

    dim3 dimBlock(16, 16);
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x, (N + dimBlock.y - 1) / dimBlock.y);

    printf("Performing warmup runs...\n");
    for (int i=0; i<3; i++){
        matrix_mul_cpu(h_A, h_B, h_C_cpu, N);
        matrix_mul_shared_mem<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);
        hipDeviceSynchronize();
    }

    memset(h_C_cpu, 0, size);
    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        matrix_mul_cpu(h_A, h_B, h_C_cpu, N);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 20.0;

    printf("Benchmarking GPU implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        hipMemset(d_C, 0, size);
        
        double start_time = get_time();
        matrix_mul_shared_mem<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }
    double gpu_avg_time = gpu_total_time / 20.0;

    printf("CPU avg time: %.2f milliseconds\n", cpu_avg_time * 1000);
    printf("GPU avg time: %.2f milliseconds\n", gpu_avg_time * 1000);
    printf("Speedup: %.2fx\n", cpu_avg_time / gpu_avg_time);
    
    hipMemcpy(h_C_gpu, d_C, size, hipMemcpyDeviceToHost);

    bool correct = true;
    int mismatches = 0;
    for (int i = 0; i < N && mismatches < 10; i++) {
        for (int j = 0; j < N && mismatches < 10; j++) {
            if (fabs(h_C_cpu[i * N + j] - h_C_gpu[i * N + j]) > 1e-3) {
                printf("Mismatch at [%d][%d]: CPU=%.6f, GPU=%.6f\n",
                       i, j, h_C_cpu[i * N + j], h_C_gpu[i * N + j]);
                mismatches++;
                correct = false;
            }
        }
    }
    
    if (correct) {
        printf("Results are correct!\n");
    } else {
        printf("Results are incorrect (%d mismatches shown)\n", mismatches);
    }
    
    free(h_A);
    free(h_B);
    free(h_C_cpu);
    free(h_C_gpu);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    
    return 0;
}